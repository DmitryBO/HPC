#include "hip/hip_runtime.h"
texture<unsigned int, 2> tex;

__global__ void kernel(unsigned int * __restrict__ image, const int M, const int N, const float sigma)
{
    const int x = threadIdx.x + blockDim.x * blockIdx.x;
    const int y = threadIdx.y + blockDim.y * blockIdx.y;

    if ((x < M) && (y < N)) {


        float c = 0;
        float s = 0;

        for (int i = x-1; i <= x+1; i++){
            for (int j = y-1; j <= y+1; j++){
                
                float pxl1 = tex2D(tex, x, y);
                float pxl2 = tex2D(tex, j, i);

                float r = exp(-pow((pxl2 - pxl1), 2) / pow(sigma, 2));
                
                float g = exp(-(pow(j - x, 2) + pow(i - y, 2)) / pow(sigma, 2));
               
                c += g*r;
                s += g*r*tex2D(tex, j, i);
            }
        }
        image[x*N + y] = s / c;
    }
}